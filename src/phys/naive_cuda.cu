#include "hip/hip_runtime.h"


// this is CUDA C++ code, so "extern "C"" is needed

extern "C" {

#include "gs_physics.h"

#include <stdio.h>
// from OpenCL
//__kernel void compute_system(__global float4 * in_p, __global float4 * g_vel, __global float4 * out_p, float4 uni_grav, int n_particles, float dt, float G_const) {

struct {

    vec4_t * GPU_in_p, *GPU_g_vel, * GPU_out_p;

} cuda_data;


#define CUDACHK(st) gpu_assert((st), __FILE__, __LINE__)

void gpu_assert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        printf("CUDA::ERROR: (at %s:%d) (code %d): %d\n", file, line, code, hipGetErrorString(code));
    }
}

#ifndef SOFT_FACTOR
#define SOFT_FACTOR 0.25
#endif


/* utils for vec4 code */

__device__ vec4_t cvec4_add(vec4_t a, vec4_t b) {
    return V4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

__device__ vec4_t cvec4_scale(vec4_t a, float b) {
    return V4(a.x*b, a.y*b, a.z*b, a.w*b);
}

__device__ float cuda_mass_to_size(float mass) {
    return cbrtf(mass);
}

__device__ vec4_t cuda_force_calc(vec4_t a, vec4_t b, float g_coef) {
    vec4_t r;
    r.x = b.x - a.x;
    r.y = b.y - a.y;
    r.z = b.z - a.z;
    float dist_sqr = r.x * r.x + r.y * r.y + r.z * r.z + SOFT_FACTOR * SOFT_FACTOR;
    float s = g_coef * a.w * b.w / dist_sqr;
    r.x *= s;
    r.y *= s;
    r.z *= s;
    r.w = 0.0;
    return r;
}


__global__ void cuda_kernel_compute_system(int n_part, vec4_t * in_p, vec4_t * g_vel, vec4_t * out_p, float G_const, vec4_t uni_grav, float dt) {
    int i = blockIdx.x*blockDim.x + threadIdx.x, j;
    if (i >= n_part) return;
    vec4_t t_force = V4(0.0, 0.0, 0.0, 0.0);
    vec4_t my_p = in_p[i], my_vel = g_vel[i];
    float my_size = cuda_mass_to_size(my_p.w);
    
    for (j = 0; j < n_part; ++j) {
      if (i != j) {
        t_force = cvec4_add(t_force, cuda_force_calc(my_p, in_p[j], G_const)); //
      }
    }
    
    t_force = cvec4_add(t_force, uni_grav);
  
    // update relevant stuff
    vec4_t new_vel = cvec4_add(my_vel, cvec4_scale(t_force, dt / my_p.w));
    vec4_t new_pos = cvec4_add(my_p, cvec4_scale(new_vel, dt));
  
    // clamping code
    if (new_pos.x + my_size > 100.0f) new_pos.x = 100.0f - my_size;
    if (new_pos.y + my_size > 100.0f) new_pos.y = 100.0f - my_size;
    if (new_pos.z + my_size > 100.0f) new_pos.z = 100.0f - my_size;
  
    if (new_pos.x - my_size < -100.0f) new_pos.x = my_size - 100.0f;
    if (new_pos.y - my_size < -100.0f) new_pos.y = my_size - 100.0f;
    if (new_pos.z - my_size < -100.0f) new_pos.z = my_size - 100.0f;
  
    out_p[i] = new_pos;
    g_vel[i] = new_vel;
}

int _plnc_hasinit = false;

void physics_loop_naive_cuda() {
    if (!_plnc_hasinit) {
        // first time initialization code
        CUDACHK(hipMalloc((void **)cuda_data.GPU_in_p, n_particles * sizeof(vec4_t)));
        CUDACHK(hipMalloc((void **)cuda_data.GPU_g_vel, n_particles * sizeof(vec4_t)));        
        CUDACHK(hipMalloc((void **)cuda_data.GPU_out_p, n_particles * sizeof(vec4_t)));        

        _plnc_hasinit = true;
    }

    CUDACHK(hipMemcpy(cuda_data.GPU_in_p, particle_data.P, n_particles * sizeof(vec4_t), hipMemcpyHostToDevice));
    CUDACHK(hipMemcpy(cuda_data.GPU_g_vel, particle_data.velocities, n_particles * sizeof(vec4_t), hipMemcpyHostToDevice));

    int local_size = 256;
    int global_size = (n_particles / local_size + ((n_particles % local_size) != 0)) * local_size;

    cuda_kernel_compute_system<<<global_size, local_size>>>(n_particles, cuda_data.GPU_in_p, cuda_data.GPU_g_vel, cuda_data.GPU_out_p, gravity_coef, universal_gravity, GS_looptime);

    CUDACHK(hipDeviceSynchronize());
    CUDACHK(hipPeekAtLastError());

    CUDACHK(hipMemcpy(particle_data.P, cuda_data.GPU_out_p, n_particles * sizeof(vec4_t), hipMemcpyDeviceToHost));
    CUDACHK(hipMemcpy(particle_data.velocities, cuda_data.GPU_g_vel, n_particles * sizeof(vec4_t), hipMemcpyDeviceToHost));


}

}

